#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>

// to build on Titan V:
//   nvcc -arch=sm_70 --ptxas-options=-v -o vanilladeriv vanilladeriv.cu;

#ifdef USE_DOUBLE
#define dfloat double
#else
#define dfloat float
#endif

#ifndef POLYNOMIAL_ORDER
#define POLYNOMIAL_ORDER 4
#endif

// note the order of the fields below is also assumed in the code.
const int _nstate = 5;

const int _R = 0, _U  = 1, _V  = 2, _W = 3, _E = 4;

const int _nvgeo = 14;
const int _XIx   = 0;
const int _ETAx  = 1;
const int _ZETAx = 2;
const int _XIy   = 3;
const int _ETAy  = 4;
const int _ZETAy = 5;
const int _XIz   = 6;
const int _ETAz  = 7;
const int _ZETAz = 8;
const int _MJ    = 9;
const int _MJI   = 10;
const int _x     = 11;
const int _y     = 12;
const int _z     = 13;

#define grav  ((dfloat) 9.81)
#define gdm1  ((dfloat) 0.4)


template <int Nq, int Np, int nvar>
  __global__ void volumerhs(dfloat * __restrict__ rhs,
                            const dfloat * __restrict__ Q,
                            const dfloat * __restrict__ vgeo,
                            const dfloat gravity,
                            const dfloat * __restrict__ D,
                            const int nelem){

  __shared__ dfloat s_D[Nq][Nq];
  __shared__ dfloat s_F[Nq][Nq][_nstate];
  __shared__ dfloat s_G[Nq][Nq][_nstate];

  dfloat r_rhsR[Nq];
  dfloat r_rhsU[Nq];
  dfloat r_rhsV[Nq];
  dfloat r_rhsW[Nq];
  dfloat r_rhsE[Nq];

  int e = blockIdx.x;
  int j = threadIdx.y;
  int i = threadIdx.x;

  s_D[j][i] = D[j*Nq+i];

#pragma unroll Nq
  for(int k=0;k<Nq;++k){
    r_rhsR[k] = 0;
    r_rhsU[k] = 0;
    r_rhsV[k] = 0;
    r_rhsW[k] = 0;
    r_rhsE[k] = 0;
  }

#pragma unroll Nq
  for(int k=0;k<Nq;++k){

    __syncthreads();

    // Load values will need into registers
    int gid = i + j*Nq + k*Nq*Nq + e*Np*_nvgeo;

    dfloat MJ = vgeo[gid + _MJ*Np];
    dfloat XIx = vgeo[gid + _XIx*Np];
    dfloat XIy = vgeo[gid + _XIy*Np];
    dfloat XIz = vgeo[gid + _XIz*Np];
    dfloat ETAx = vgeo[gid + _ETAx*Np];
    dfloat ETAy = vgeo[gid + _ETAy*Np];
    dfloat ETAz = vgeo[gid + _ETAz*Np];
    dfloat ZETAx = vgeo[gid + _ZETAx*Np];
    dfloat ZETAy = vgeo[gid + _ZETAy*Np];
    dfloat ZETAz = vgeo[gid + _ZETAz*Np];
    dfloat z = vgeo[gid +  _z*Np];


    int qid = i + j*Nq + k*Nq*Nq + e*Np*nvar;

    dfloat R = Q[qid + _R*Np];
    dfloat U = Q[qid + _U*Np];
    dfloat V = Q[qid + _V*Np];
    dfloat W = Q[qid + _W*Np];
    dfloat E = Q[qid + _E*Np];

    dfloat P = gdm1*(E - (U*U + V*V + W*W)/(2*R) - R*gravity*z);

    dfloat Rinv = 1 / R;

    dfloat fluxR_x = U;
    dfloat fluxU_x = Rinv * U * U + P;
    dfloat fluxV_x = Rinv * U * V;
    dfloat fluxW_x = Rinv * U * W;
    dfloat fluxE_x = Rinv * U * (E + P);

    dfloat fluxR_y = V;
    dfloat fluxU_y = Rinv * V * U;
    dfloat fluxV_y = Rinv * V * V + P;
    dfloat fluxW_y = Rinv * V * W;
    dfloat fluxE_y = Rinv * V * (E + P);

    dfloat fluxR_z = W;
    dfloat fluxU_z = Rinv * W * U;
    dfloat fluxV_z = Rinv * W * V;
    dfloat fluxW_z = Rinv * W * W + P;
    dfloat fluxE_z = Rinv * W * (E + P);

    s_F[i][j][ _R] = MJ * (XIx * fluxR_x + XIy * fluxR_y + XIz * fluxR_z);
    s_F[i][j][ _U] = MJ * (XIx * fluxU_x + XIy * fluxU_y + XIz * fluxU_z);
    s_F[i][j][ _V] = MJ * (XIx * fluxV_x + XIy * fluxV_y + XIz * fluxV_z);
    s_F[i][j][ _W] = MJ * (XIx * fluxW_x + XIy * fluxW_y + XIz * fluxW_z);
    s_F[i][j][ _E] = MJ * (XIx * fluxE_x + XIy * fluxE_y + XIz * fluxE_z);

    s_G[i][j][ _R] = MJ * (ETAx * fluxR_x + ETAy * fluxR_y + ETAz * fluxR_z);
    s_G[i][j][ _U] = MJ * (ETAx * fluxU_x + ETAy * fluxU_y + ETAz * fluxU_z);
    s_G[i][j][ _V] = MJ * (ETAx * fluxV_x + ETAy * fluxV_y + ETAz * fluxV_z);
    s_G[i][j][ _W] = MJ * (ETAx * fluxW_x + ETAy * fluxW_y + ETAz * fluxW_z);
    s_G[i][j][ _E] = MJ * (ETAx * fluxE_x + ETAy * fluxE_y + ETAz * fluxE_z);

    dfloat r_HR = MJ * (ZETAx * fluxR_x + ZETAy * fluxR_y + ZETAz * fluxR_z);
    dfloat r_HU = MJ * (ZETAx * fluxU_x + ZETAy * fluxU_y + ZETAz * fluxU_z);
    dfloat r_HV = MJ * (ZETAx * fluxV_x + ZETAy * fluxV_y + ZETAz * fluxV_z);
    dfloat r_HW = MJ * (ZETAx * fluxW_x + ZETAy * fluxW_y + ZETAz * fluxW_z);
    dfloat r_HE = MJ * (ZETAx * fluxE_x + ZETAy * fluxE_y + ZETAz * fluxE_z);

    // one shared access per 10 flops
#pragma unroll Nq
    for(int n=0;n<Nq;++n){
      dfloat  Dkn = s_D[k][n];

      r_rhsR[n] += Dkn * r_HR;
      r_rhsU[n] += Dkn * r_HU;
      r_rhsV[n] += Dkn * r_HV;
      r_rhsW[n] += Dkn * r_HW;
      r_rhsE[n] += Dkn * r_HE;
    }

    r_rhsW[k] -= MJ * R * gravity;

    __syncthreads();

    // loop of XI-grid lines
#pragma unroll Nq
    for(int n=0;n<Nq;++n){
      dfloat Dni = s_D[n][i];
      dfloat Dnj = s_D[n][j];

      r_rhsR[k] += Dni * s_F[n][j][_R];
      r_rhsR[k] += Dnj * s_G[i][n][_R];

      r_rhsU[k] += Dni * s_F[n][j][_U];
      r_rhsU[k] += Dnj * s_G[i][n][_U];

      r_rhsV[k] += Dni * s_F[n][j][_V];
      r_rhsV[k] += Dnj * s_G[i][n][_V];

      r_rhsW[k] += Dni * s_F[n][j][_W];
      r_rhsW[k] += Dnj * s_G[i][n][_W];

      r_rhsE[k] += Dni * s_F[n][j][_E];
      r_rhsE[k] += Dnj * s_G[i][n][_E];
    }
  }

#pragma unroll Nq
  for(int k=0;k<Nq;++k){
    int gid = i + j*Nq + k*Nq*Nq + e*Np*_nvgeo;
    dfloat MJI = vgeo[gid +  _MJI*Np];

    int qid = i + j*Nq + k*Nq*Nq + e*Np*nvar;

    rhs[qid+_U*Np] += MJI*r_rhsU[k];
    rhs[qid+_V*Np] += MJI*r_rhsV[k];
    rhs[qid+_W*Np] += MJI*r_rhsW[k];
    rhs[qid+_R*Np] += MJI*r_rhsR[k];
    rhs[qid+_E*Np] += MJI*r_rhsE[k];
  }
}

void randArray(int N, dfloat base, dfloat range, dfloat **q, dfloat **c_q){

  *q = (dfloat*) calloc(N, sizeof(dfloat));
  hipMalloc(c_q, N*sizeof(dfloat));

  for(int n=0;n<N;++n){
    q[0][n] = base + drand48()*range;
  }

  hipMemcpy(c_q[0], q[0], N*sizeof(dfloat), hipMemcpyHostToDevice);

}

int main(int argc, char **argv){

  srand48(1234);

  const int N = POLYNOMIAL_ORDER;
  const int nelem = 4000;

  const int Nq = N+1;
  const int Np = Nq*Nq*Nq;

  const int Ntotal = Np*nelem*_nstate;

  dfloat *Q, *c_Q;
  randArray(Ntotal, 0., 1., &Q, &c_Q);

  for(int e=0;e<nelem;++e){
    for(int n=0;n<Np;++n){
      int idR = n + _R*Np + e*_nstate*Np;
      int idE = n + _E*Np + e*_nstate*Np;

      Q[idR] += 2.;
      Q[idE] += 20.;

    }
  }

  hipMemcpy(c_Q, Q, nelem*_nstate*Np*sizeof(dfloat), hipMemcpyHostToDevice);

  const int Gtotal = Np*nelem*_nvgeo;

  dfloat *vgeo, *c_vgeo;
  randArray(Gtotal, 0, 1., &vgeo, &c_vgeo);

  // Make sure the entries of the mass matrix satisfy the inverse relation
  for(int e=0;e<nelem;++e){
    for(int n=0;n<Np;++n){
      int idMJ = n + _MJ*Np + e*_nvgeo*Np;
      int idMJI = n + _MJI*Np + e*_nvgeo*Np;

      vgeo[idMJ] += 3;
      vgeo[idMJI] = 1./vgeo[idMJ];

    }
  }
  hipMemcpy(c_vgeo, vgeo, nelem*_nvgeo*Np*sizeof(dfloat), hipMemcpyHostToDevice);

  dfloat *D, *c_D;
  randArray(Nq*Nq, 1., 1., &D, &c_D);

  dfloat *rhs, *c_rhs;

  srand48(1234);
  randArray(Ntotal, 1., 1., &rhs, &c_rhs);

  dim3 G(nelem,1,1);
  dim3 B2(Nq,Nq,Nq);
  dim3 B3(Nq,Nq,1);

  volumerhs<Nq, Np, _nstate> <<< G, B3 >>> (c_rhs, c_Q, c_vgeo, grav, c_D, nelem);

  hipDeviceSynchronize();

  exit(0);
  return 0;
}
